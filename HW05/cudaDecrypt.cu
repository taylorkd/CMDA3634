#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ unsigned int modprodC(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExpC(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodC(aExpb, z, p);
    z = modprodC(z, z, p);
    b /= 2;
  }
  return aExpb;
}


__global__ void findX(unsigned int p, unsigned int g, unsigned int h, unsigned int *x)
{
//unsigned int block = blockIdx.x;
//unsigned int blocksize = blockDim.x;
//unsigned int thread = threadIdx.x;
//unsigned int id=thread + block*blocksize;
if (*x==0 || modExpC(g,*x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExpC(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        *x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }


}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */




  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
FILE *pub_key = fopen("public_key.txt","r");
FILE *cyperT = fopen("message.txt","r");
fscanf(pub_key,"%u\n%u\n%u\n%u",&n,&p,&g,&h);
fclose(pub_key);
fscanf(cyperT,"%u\n",&Nints);
unsigned int *a=(unsigned int *) malloc(Nints*sizeof(unsigned int));
unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
for(unsigned int i=0;i<Nints;i++)
{ 
 fscanf(cyperT,"%u %u\n",&Zmessage[i],&a[i]);
 
}
fclose(cyperT); 
  // find the secret key
unsigned int Nthreads = Nints;
unsigned int Nblocks = (n+Nthreads-1)/Nthreads;
hipMalloc((void**)&x,1*sizeof(unsigned int));
printf("%u\n",x);
findX<<< Nthreads,Nblocks >>>(p,g,h,&x);
//hipDeviceSynchronize();
printf("x:%u\n",x);
unsigned int foundx;
  /* Q3 After finding the secret key, decrypt the message */
hipMemcpy(&x,&foundx,1*sizeof(unsigned int),hipMemcpyHostToDevice);
printf("secret key:%u\n",foundx);
ElGamalDecrypt(Zmessage,a,Nints,p,foundx);
unsigned char *message = (unsigned char *) malloc(Nints*sizeof(unsigned char));
unsigned int charsPerInt = (n-1)/8;
unsigned int Nchars = Nints*charsPerInt;
convertZToString(Zmessage,Nints,message,Nchars);
printf("Decrypted Message = \"%s\"\n",message);


hipFree(&x);
  return 0;

 
}
