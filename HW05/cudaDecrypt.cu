#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__global__ void findX(unsigned int p, unsigned int g, unsigned int h, unsigned int *x)
{
unsigned int block = blockIdx.x;
unsigned int blocksize = blockDim.x;
unsigned int thread = threadIdx.x;
unsigned int id=thread + block*blocksize;
if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }


}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */




  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */


unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
FILE *pub_key = fopen("public_key.txt","r");
FILE *cyperT = fopen("message.txt","r");
fscanf(pub_key,"%u\n%u\n%u\n%u",&n,&p,&g,&h);
fclose(pub_key);
fscanf(cyperT,"%u\n",&Nints);
for(unsigned int i=0;i<Nints;i++)
{ 
 fscanf(cyperT,"%u %u\n",&Zmessage[i],&a[i]);
 
}
fclose(cyperT); 
  // find the secret key
unsigned int Nthreads = Nints;
unsigned int Nblocks = (n+Nthreads-1)/Nthreads

findx<<< Nthreads,Nblocks >>>(p,g,h,x);
hipDeviceSynchronize();

  /* Q3 After finding the secret key, decrypt the message */
hipMalloc(&x,Nints*sizeof(unsigned int));
hipMemcpy(x,x,Nints*sizeof(unsigned int),hipMemcpyHostToDevice);
ElGamalDecrypt(Zmessage,a,Nints,p,x);
int bufferSize = 1024;
unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
unsigned int charsPerInt = (n-1)/8;
printf("%u\n",charsPerInt);
unsigned int Nchars = Nints*charsPerInt;
convertZToString(Zmessage,Nints,message,Nchars);
printf("Decrypted Message = \"%s\"\n",message);

free(x);
hipFree(x);
  return 0;

 
}
